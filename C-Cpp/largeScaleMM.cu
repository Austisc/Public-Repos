#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h> 
#include <iostream>
#include <omp.h>
/*

Kernel Function for matrix initialisation
@matrix int* address of matrix
@numRows int number of rows
@numCOls int number of cols

*/

__global__ void MatInit(int* matrix, int numRows, int numCols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    //row and col for 1D Mat
    if (row < numRows && col < numCols) { //so not all will have have been initliase
        matrix[row * numCols + col] = row*col;//initialise as 1
    }
}
/*

Kernel Function to transpose matrix
@matrix int* address of input
@matrix int* address of output
@numRows int number of rows
@numCols int number of cols

*/
__global__ void TransposeKernel(const int* input, int* output, int numRows, int numCols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x; //index of row
    int col = blockIdx.y * blockDim.y + threadIdx.y;//index of col

    if (row < numRows && col < numCols) {
        output[row * numCols + col] = input[col * numRows + row]; //to transpose
    }
}

/*
Host Function for matrix initialisation

@numRows int number of rows
@numCOls int number of cols
@return matrix int* 
*/

int* InitializeMatrix(int numRows, int numCols) {
    int* c_matrix;
    int* new_matrix = (int*)malloc(numRows * numCols * sizeof(int));
    // allocate device memory for devices
    hipMalloc((void**)&c_matrix, numRows * numCols * sizeof(int));

    // define thread block and grid dimension
    dim3 dimBlock(16, 16);
    dim3 gridSize((numCols + dimBlock.x - 1) / dimBlock.x, (numRows + dimBlock.y - 1) / dimBlock.y); //using dimBlock to determine gridSize

    // launch kernel to initialize the matrix on the device
    MatInit << <gridSize, dimBlock >> > (c_matrix, numRows, numCols);


    // copy matrix generate by kernel function MatInit from device to host
    hipMemcpy(new_matrix, c_matrix, numRows * numCols * sizeof(int), hipMemcpyDeviceToHost);
    // free device memory
    hipFree(c_matrix);
    return new_matrix;

}


/*
Host Function for matrix transpose
using kernel function to transpose
@numRows int number of rows
@numCOls int number of cols
@return matrix int*
*/


int* Transpose(int* Mat, int numRows, int numCols) {


    int* d_input = NULL;
    int* d_output = NULL;
    int* output = (int*)malloc(numRows * numCols * sizeof(int));

    // allocate device memory for input and output matrix
    hipMalloc((void**)&d_input, numRows * numCols * sizeof(int));
    hipMalloc((void**)&d_output,numRows *numCols * sizeof(int)); 

    // copy input matrix for transpose from host to device
    hipMemcpy(d_input,Mat,numRows * numCols * sizeof(int), hipMemcpyHostToDevice);

    // define thread block and grid dimensions
    dim3 dimBlock(16,16);
    dim3 gridSize((numCols + dimBlock.x - 1) / dimBlock.x, (numRows + dimBlock.y - 1) / dimBlock.y); //using dimBlock to determine gridSize
    

    // launch the transpose kernel function
    TransposeKernel << <gridSize, dimBlock >> > (d_input, d_output, numRows, numCols);

    // copy the transposed matrix from device to host
    hipMemcpy(output, d_output, numRows * numCols * sizeof(int), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_input);
    hipFree(d_output);
    //return the transposed matrix
    return output;
}
//constant for L and G
#define BLOCKSIZE 128

#define L 32
#define G 32
/*
Kernel Function for matrix Multiplicaiton
it deal with sub-grpup of matrix deifned by LxL which it will produce LxLxG matrix
thus, it will only deal with sizes of matrices that is multiplicaiton of 32 usign the techniques from research paper
//Step I: global memory reduction
//Step II : shared memory reductions
@matA int* MatrixA
@matB int* MatrixB
@matC int* MatrixC
@numRows int number of rows
@numCOls int number of cols
@N the original size of Matrix 
sources: https://ieeexplore.ieee.org/abstract/document/5704475
*/

__global__ void MMKernel(int* MatA, int* MatB, int* MatC, int numRows, int numCols, int N) {
        //Coalesced global memory asses
        // it is all loaded from globl memory, Each thread will be sharing the same block
        // which it can lead to On-fly computing which it  while reading , it will not copy the local memory,
        // but accumlates as a partial sum at PS __shared__ blocksize which for each thread
        __shared__ int PS[BLOCKSIZE];
        unsigned int i = blockIdx.z *(BLOCKSIZE) + threadIdx.x;
        unsigned int tid = threadIdx.x;
        unsigned int gridSize = BLOCKSIZE * gridDim.z;

        PS[tid] = 0;

        // Step I: global memory reduction 
    
        while (i < N) {
            
            PS[tid] += MatA[blockIdx.x * L + i] * MatB[blockIdx.x * L + i]; // compute the sharing 
            i += gridSize;
        }

 
    __syncthreads();

    // Step 2: shared memory reduction
    //  Bank conflitc t free shared memory usage.
        //since there is an shared memory, parallel reduction flow can avoids the bank conflicts , which across the conflict shared memory 
        // by making threads to share the memory, which half of the threads can immediatelly access the result from the shared memory.
        // 
        // 
         //Loop Unrolling 
         // using reverse binary tree flow without for loop since GPU for loop has efficinecy and since we will use 2^N which =< BlockSize
         // 
         // 
         // removing number of threads synchronization , which does not exceed 32, so synthread is removed for the last iteration
         // since active threads during last 6 iteration is inside the same wrap
         // 
         // 
         // 
         // 
        //since it is outside the wrap ,for threads id >64 so, __synthread() is needed
    
        if (BLOCKSIZE >= 256) {
            if (tid < 128) {
                PS[tid] += PS[tid + 128];
            }
            __syncthreads();
        }
        if (BLOCKSIZE >= 128) {
            if (tid < 64) {
                PS[tid] += PS[tid + 64];
            }
            __syncthreads();
        }
        //sicne it is inside the wrap for threads id =<64 so, __synthread() is not needed
        if (tid < 32) {
            if (BLOCKSIZE >= 64) { PS[tid] += PS[tid + 32]; 
            }
            if (BLOCKSIZE >= 32) { PS[tid] += PS[tid + 16];
            }
            if (BLOCKSIZE >= 16) { PS[tid] += PS[tid + 8]; 
            }
            if (BLOCKSIZE >= 8) { PS[tid] += PS[tid + 4]; 
            }
            if (BLOCKSIZE >= 4) { PS[tid] += PS[tid + 2];  
            }
            if (BLOCKSIZE >= 2) { PS[tid] += PS[tid + 1];
            }
           
        }
        // it will sub the last index
        if (tid == 0) {
            int index = blockIdx.x*L*G + blockIdx.y*G + blockIdx.z ;
            MatC[index] = PS[tid];
        }
        __syncthreads();
    }


/*
Host  Function for matrix Multiplicaiton
it will partition matrix into subgroup and then the subgroup matrix will be done by sub-group matrix multiplcation
and then subgroup will be copy to the corresponding address of Matrix C as intended
Note: only work for  32 of NxN matrix with multiplicaiton of 32 

@matA int* MatrixA
@matB int* MatrixB
@numRows int number of rows
@numCOls int number of cols
@ return matC int* MatrixC
*/


int*  matrixMM(int* MatA, int* MatB, unsigned int numRows, unsigned int numCols) {

        int* subMatA = NULL;
        int* subMatB = NULL;
        int* subMatC = NULL;

        //initialise all the matrix
        //allocate memory
        int* MatC = (int*)malloc(numRows * numCols  * sizeof(int));
        
       
        // partition it into N/L X N/L groups with each subgroup got size of LxL matrixes
        for (unsigned int i = 0; i < numRows / L; i++) 
            for (unsigned int j = 0; j < numCols / L; j++) {

              
                subMatA = (int*)malloc(L * numRows * sizeof(int)); // it will have L xN subgroup of Matrix A for subgroup MM
                subMatB = (int*)malloc(L * numCols * sizeof(int)); // it will have L xN subgroup of Matrix B for subgroup MM
                subMatC = (int*)malloc(L * L * G * sizeof(int));// it will have L xL xG subgroup of Matrix C using kernel function MM

                // partition sub-group of Mat A and Mat B from i×L to ((i+1)×L-1) but respresent into 1D array 
                //using #pragma to improve the intilialisation of subMatrix
                #pragma omp parallel for collapse(2)
                for (unsigned int row = 0; row < L; row++) {
                    for (unsigned int col = 0; col < numCols; col++) {
                        unsigned int ind = (i * L + row) * numCols + col;

                        if (ind < numRows * numCols) {
                            subMatA[row * numCols + col] = MatA[ind]; 
                            subMatB[row * numCols + col] = MatB[ind];
                        }
                    }
                }
                int* c_MatA = NULL;
                int* c_MatB = NULL;
                int* c_MatC = NULL;
               
                hipMalloc((void**)&c_MatA, L * numRows * sizeof(int));
                hipMalloc((void**)&c_MatB, L * numCols * sizeof(int));
                hipMalloc((void**)&c_MatC, L * L * G * sizeof(int));
                hipMemset(c_MatA, 0, L * numRows * sizeof(int));
                hipMemset(c_MatB, 0, L * numCols * sizeof(int)); //set all index in devices to 0
                hipMemset(c_MatC, 0, L * L * G * sizeof(int)); //set all index in devices to 0
               
                hipMemcpy(c_MatA, subMatA, L * numRows * sizeof(int), hipMemcpyHostToDevice); //copy to the device memory
                hipMemcpy(c_MatB, subMatB, L * numCols * sizeof(int), hipMemcpyHostToDevice); //copy to the device memory

                // define thread block and grid dimensions
                dim3 dimBlock(BLOCKSIZE, 1, 1); //BlockSize will be using size 256
                dim3 gridSize(L, L, G); // it will initialise gridSize LxLxG which the same for matrix output


                //laucnh the Matrix Multiplcation kernal function
                MMKernel << <gridSize, dimBlock >> > (c_MatA, c_MatB, c_MatC, L, L, numRows);
                hipMemcpy(subMatC, c_MatC, L * L * G * sizeof(int), hipMemcpyDeviceToHost); //copy from device to host

                int N = numRows;
               
                //reduce each partial result according to a and b which is respective to i and j and reduce to a single value for Mat(i,j)
                int sum;
                //using omp to improve parallelisaiton with reduction fo the sum
                #pragma omp parallel for collapse(2) redunction(+:sum)
                for (unsigned int a = 0; a < L; a++) {
                    for ( unsigned int b = 0; b < L; b++) {
                        int sum = 0;
                        for (unsigned int c = 0; c < G; c++) {
                            sum += subMatC[a *(L )* G + b * G + c];//reduce the partial result to a single Value

                        }
                        
                        unsigned int index = (i * L + a) * numCols + (j*L+ b);//index to assign to MatC
                        /*printf("Index: %d\n", index);*/
                        MatC[index] = sum;
                       
                    }
                }
                hipFree(c_MatA);
                hipFree(c_MatB);
                hipFree(c_MatC);
            }
        //free all memory
       
        free(subMatA);
        free(subMatB);
        free(subMatC);

        return MatC;
    }




int main(int argc, char* argv[]) {
    const unsigned int numRows = pow(2,10); //  as long as it is multiplicaiton of L and power of 2
    const unsigned int numCols = pow(2,10); // as long as it is multiplicaiton of L  and power of 2

    // call the function to initialize the matrix
    int* h_matrix = InitializeMatrix(numRows, numCols);  // initialise Matrix A
    int* g_matrix = InitializeMatrix(numRows, numCols); //initialise Matrix B 
    int* t_matrix = Transpose(g_matrix, numRows, numCols); //tranpose  Matrix B 

    clock_t start_timeA = clock();

    //Naive Multiplication
    int* MatC = (int*)malloc(numCols * numRows * sizeof(int));
    for (int i = 0; i < numRows; i++) {
        for (int j = 0; j < numCols; j++) {
            MatC[i * numCols + j] = 0;
            for (int k = 0; k < numCols; k++) {
                MatC[i * numCols + j] += h_matrix[i * numCols + k] * g_matrix[k * numCols + j];
            }
        }
    }
    clock_t end_timeA = clock();// end time
    double elapsed_secondAs = static_cast<double>(end_timeA - start_timeA) / CLOCKS_PER_SEC; // time finish MM
    
    clock_t start_time = clock();
    //Matrix Multiplication calculation
    int* MatCC = matrixMM(h_matrix, t_matrix, numRows, numCols);
    //for lop to indicates the correctness of the matrix since both are 1 matrices in my cases, each Mat(i,j) will equal to N
    clock_t end_time = clock();// end time
    double elapsed_seconds = static_cast<double>(end_time - start_time) / CLOCKS_PER_SEC; // time finish MM
    int br = 0;
   
    for (int i = 0; i < numRows; i++) {
        int br = 0;
        for (int j = 0; j < numCols; j++) {
           
            int index = i * numCols + j;
          
            if (MatCC[index] !=  MatC[index]) {
                br = 1;
                //printf(" % d, % d ", i, j);
                break;
               
            }
        }
        if (br) {
            break;
        }

      
        
    }

 
    if((br==0)) {// if it is completed, it will indicates time
        std::cout << "Elapsed time of  Naive Matrix Multiplication : " << elapsed_secondAs << " seconds\n" << std::endl;
        std::cout << "Elapsed time of  CUDA Matrix Multiplication : " << elapsed_seconds << " seconds\n" << std::endl;
    }
    else { // if it is bad, it will not indicates time
        std::cout << "Else , It is incorrect "<< std::endl; // if there is an error
    }
    free(g_matrix);
    free(MatCC);
    return 0;
}
